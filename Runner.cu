#include "Runner.cuh"
#include <hiprand.h>
#include <cstdlib>
Runner::Runner(int id) : id(id), position(0), speed(0)
{

}

Runner::Runner()
{
    id = 0;
    position = 0;
    speed = 0;
}

int Runner::getSpeed() const
{
    return speed;
}

int Runner::getId() const
{
    return id;
}

int Runner::getPosition() const
{
    return position;
}

void Runner::move()
{
    position += speed;
}

void Runner::incrementId()
{
    id++;
}

void Runner::updateSpeed()
{
    int newSpeed = (rand() % 5) + 1;
    this->speed = newSpeed;
}
