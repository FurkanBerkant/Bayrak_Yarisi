// Runner.cu kaynak dosyas�
#include "Team.cuh"
#include <hiprand.h>
#include <cstdlib>

Team::Team(int id, int numRunners) : id(id) {
    for (int i = 0; i < numRunners; i++) {
        runners.push_back(Runner(i + 1));
    }
}


std::vector<Runner>& Team::getRunners()
{
    return runners;
}

void Team::run()
{
    for (int i = 0; i < runners.size(); i++) {
        runners[i].move();
    }
}


int Team::getId() const
{
    return id;
}
