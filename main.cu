#include <iostream>
#include <vector>
#include <sstream>
#include <chrono>
#include <thread>
#include "Runner.cuh"
#include "Team.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cu"
#include "hiprand.h"

const int TAKIM_SAYISI = 400;
const int KOSUCU_SAYISI = 4;
const int YARIS_UZUNLUGU = 400;
const int BITIS_NOKTASI = YARIS_UZUNLUGU;
const int NUM_BLOCKS = 400;
const int THREADS_PER_BLOCK = 4;

int main() {
    std::srand(static_cast<unsigned>(std::time(nullptr)));
    std::vector<Team*> teams;
    hipError_t cudaStatus = hipSuccess;
    for (int i = 0; i < TAKIM_SAYISI; ++i) {
        Team* team = new Team(i + 1, KOSUCU_SAYISI);
        teams.push_back(team);
    }


    int time_elapsed = 0;
    bool race_finished = false;

    std::cout << "Hangi takimlarin bilgilerini gormek istersiniz? (Ornek: '1 3'): ";
    std::string secilen_takimlar;
    std::getline(std::cin, secilen_takimlar);

    std::vector<int> secilen_takimlar_indeks;
    std::istringstream iss(secilen_takimlar);
    int takim_no;

    while (iss >> takim_no) {
        takim_no--;
        if (takim_no >= 0 && takim_no < TAKIM_SAYISI) {
            secilen_takimlar_indeks.push_back(takim_no);
        }
    }
    std::cout << "Yaris basladi!" << std::endl;
    Runner* d_runners;
    hipMalloc((void**)&d_runners, TAKIM_SAYISI * KOSUCU_SAYISI * sizeof(Runner));
    hiprandState_t* d_states;
    hipMalloc((void**)&d_states, TAKIM_SAYISI * KOSUCU_SAYISI * sizeof(hiprandState_t));

    int seed = time(NULL);
    initCurand <<<NUM_BLOCKS, THREADS_PER_BLOCK >>> (d_states, seed);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "initCurand launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    updateSpeedKernel << <NUM_BLOCKS, THREADS_PER_BLOCK >> > (d_runners, d_states);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "updateSpeedKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    while (!race_finished) {

        std::cout << "Zaman: " << time_elapsed << " saniye" << std::endl;
        for (int i = 0; i < secilen_takimlar_indeks.size(); i++) {
            int teamIndex = secilen_takimlar_indeks[i];
            teams[teamIndex]->run();
            std::cout << "Takim " << teamIndex + 1 << ":" << std::endl;

            bool all_runners_finished = true;
            for (int j = 0; j < KOSUCU_SAYISI; j++) {
                Runner& runner = teams[teamIndex]->getRunners()[j];
                //d_runners[teamIndex * KOSUCU_SAYISI + j].updateSpeedGPU(&d_states[teamIndex * KOSUCU_SAYISI + j]);
                runner.move();
                if (runner.getPosition() % 100 == 0 && runner.getPosition() > 0) {
                    runner.incrementId();
                }
                std::cout << "   Kosucu " << runner.getId()
                    << ": Pozisyon = " << runner.getPosition()
                    << "m, Hiz = " << runner.getSpeed() << "m/s" << std::endl;
                if (runner.getPosition() <= BITIS_NOKTASI) {
                    all_runners_finished = false;
                    break;
                }
            }
            if (all_runners_finished) {
                race_finished = true;
            }
            std::this_thread::sleep_for(std::chrono::milliseconds(500));
        }
        time_elapsed++;
    }
    for (int i = 0; i < TAKIM_SAYISI; i++) {
        delete teams[i];
    }
Error:
    hipFree(d_runners);
    hipFree(d_states);

    return 0;
}
