#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "Runner.cuh"


__global__ void updateSpeedKernel(Runner* runners, hiprandState_t* states) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    runners[index].updateSpeedGPU(&states[index]);
}

__global__ void initCurand(hiprandState_t* states, unsigned long long seed) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, index, 0, &states[index]);
}
__global__ void myKernel(Runner* d_runners, hiprandState_t* d_states, int teamIndex) {
    int j = threadIdx.x; // GPU iş parçacığının indeksi

    d_runners[teamIndex * 4 + j].updateSpeedGPU(&d_states[teamIndex * 4 + j]);
}